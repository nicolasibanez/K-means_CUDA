#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// k-means on CPU & GPU
// S. Vialle March 2022 (with the help of G. He)
////////////////////////////////////////////////////////////////////////////////

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h> 
#include <hip/hip_runtime.h>
#include <hipblas.h>              // for CUBLAS_GEAM
#include <hiprand/hiprand_kernel.h>          // for CURAND_UNIFORM
#include <float.h>

#include "main.h"
#include "init.h"
#include "kmeans_gpu.h"


/*----------------------------------------------------------------------------*/
/* Define pointers on GPU variables, and GPU symbols                          */
/*----------------------------------------------------------------------------*/
// Choose the ONE you need (big array)
T_real *GPU_instance;      //[NB_INSTANCES][NB_DIMS] --> [NB_INSTANCES * NB_DIMS]
T_real *GPU_instance_T;    //[NB_DIMS][NB_INSTANCES] --> [NB_DIMS * NB_INSTANCES]

// Choose the one you need or both (small arrays)
T_real *GPU_centroid;      //[NB_CLUSTERS][NB_DIMS] --> [NB_CLUSTERS * NB_DIMS]
T_real *GPU_centroid_T;    //[NB_DIMS][NB_CLUSTERS] --> [NB_DIMS * NB_CLUSTERS]

int *GPU_label;             //[NB_INSTANCES] Label of each point
int *GPU_change;            //[NB_INSTANCES] Flag recording the change of label
int *GPU_count;             //[NB_CLUSTERS]  Count of instance points in each cluster
int *GPU_failed;

__device__ unsigned long long GPU_change_total; // nb of label changes at current iter
unsigned long long *AdrGPU_change_total = NULL;

hiprandState *devStates;     // To use hiprand
hipblasHandle_t cublasHandle;// To activate cublas (speedup the GPU!)


/*-------------------------------------------------------------------------------*/
/* Init and finalize the GPU device.                                             */
/*-------------------------------------------------------------------------------*/
void gpu_Init()
{
  hipInit(0);

  // Allocate memory space for GPU arrays
  
  // Choose ONE (Big array)
  CHECK_CUDA_SUCCESS(hipMalloc((void**) &GPU_instance, sizeof(T_real)*NB_INSTANCES*NB_DIMS), "Dynamic allocation for GPU_instance");
  CHECK_CUDA_SUCCESS(hipMalloc((void**) &GPU_instance_T, sizeof(T_real)*NB_DIMS*NB_INSTANCES), "Dynamic allocation for GPU_instance_T");
  

  // Choose one or both (small array)
  CHECK_CUDA_SUCCESS(hipMalloc((void**) &GPU_centroid, sizeof(T_real)*NB_CLUSTERS*NB_DIMS), "Dynamic allocation for GPU_centroid");
  CHECK_CUDA_SUCCESS(hipMalloc((void**) &GPU_centroid_T, sizeof(T_real)*NB_DIMS*NB_CLUSTERS), "Dynamic allocation for GPU_centroid_T");
  
  CHECK_CUDA_SUCCESS(hipMalloc((void**) &GPU_label, sizeof(int)*NB_INSTANCES), "Dynamic allocation for GPU_label");
  CHECK_CUDA_SUCCESS(hipMalloc((void**) &GPU_change, sizeof(int)*NB_INSTANCES), "Dynamic allocation for GPU_change");
  CHECK_CUDA_SUCCESS(hipMalloc((void**) &GPU_count, sizeof(int)*NB_CLUSTERS), "Dynamic allocation for GPU_count");

  CHECK_CUDA_SUCCESS(hipMalloc((void**) &GPU_failed, sizeof(int)*1), "Dynamic allocation for GPU_failed");
 
  // Initialize an array of "hiprandState) for using hiprand
  CHECK_CUDA_SUCCESS(hipMalloc((void**) &devStates, sizeof(hiprandState)*NB_CLUSTERS), "Dynamic allocation for devStates");

  // Get address of GPU symbols
  CHECK_CUDA_SUCCESS(hipGetSymbolAddress((void **) &AdrGPU_change_total, GPU_change_total), 
                     "Get the address of GPU_change_total");
                     
  // Initialize CUBLAS lib usage
  CHECK_CUBLAS_SUCCESS(hipblasCreate(&cublasHandle), "Init of the CUBLAS lib handle"); 
}

void gpu_Finalize()
{
  // Free dynamic allocations (function of the arrays you used)
  CHECK_CUDA_SUCCESS(hipFree(GPU_instance), "Free the dynamic allocation for GPU_instance");
  CHECK_CUDA_SUCCESS(hipFree(GPU_instance_T), "Free the dynamic allocation for GPU_instance_T");

  CHECK_CUDA_SUCCESS(hipFree(GPU_centroid), "Free the dynamic allocation for GPU_centroid");
  CHECK_CUDA_SUCCESS(hipFree(GPU_centroid_T), "Free the dynamic allocation for GPU_centroid_T");
  CHECK_CUDA_SUCCESS(hipFree(GPU_label), "Free the dynamic allocation for GPU_label");
  CHECK_CUDA_SUCCESS(hipFree(GPU_change), "Free the dynamic allocation for GPU_change");
  CHECK_CUDA_SUCCESS(hipFree(GPU_count), "Free the dynamic allocation for GPU_count");

  CHECK_CUDA_SUCCESS(hipFree(GPU_failed), "Free the dynamic allocation for GPU_failed");
  
  // Free array of curandStates
  CHECK_CUDA_SUCCESS(hipFree(devStates), "Free the dynamic allocation for devStates");

  // Free CUBLAS lib usage
  CHECK_CUBLAS_SUCCESS(hipblasDestroy(cublasHandle), "Free the CUBLAS lib");
}


/*-------------------------------------------------------------------------------*/
/* Transfer of CPU input data into GPU symbols                                   */
/*-------------------------------------------------------------------------------*/
void gpu_SetDataOnGPU()
{
  // Transfer instance[] or instance_T[] .... as you want                  // TO DO
  CHECK_CUDA_SUCCESS(hipMemcpy(GPU_instance, instance,
                                sizeof(T_real)*NB_DIMS*NB_INSTANCES, 
                                hipMemcpyHostToDevice),
                      "Transfer instance...");


  
  T_real alpha = 1.0f;
  T_real beta = 0.0f;
  CHECK_CUBLAS_SUCCESS(CUBLAS_GEAM(cublasHandle,
                                  HIPBLAS_OP_T, HIPBLAS_OP_N,
                                  NB_INSTANCES, NB_DIMS,
                                  &alpha, GPU_instance, NB_DIMS,
                                  &beta, NULL, NB_INSTANCES,
                                  GPU_instance_T, NB_INSTANCES), 
                      "Use CUBLAS_GEAM to transpose GPU_instance");
}


/*-------------------------------------------------------------------------------*/
/* Transfer of GPU results into CPU array                                        */
/*-------------------------------------------------------------------------------*/
void gpu_GetResultOnCPU()
{
  // Transfer labels computed on GPU, to the CPU                           // TO DO
  CHECK_CUDA_SUCCESS(hipMemcpy(label, GPU_label, 
                               sizeof(int)*NB_INSTANCES, 
                               hipMemcpyDeviceToHost),
                    "Transfer labels...");

  // Transfer final centroids computed on GPU, to the CPU                  // TO DO
  // MIGHT NEED TO TRANSPOSE !

  T_real alpha = 1.0f;
  T_real beta = 0.0f;
  CHECK_CUBLAS_SUCCESS(CUBLAS_GEAM(cublasHandle,
                                  HIPBLAS_OP_T, HIPBLAS_OP_N,
                                  NB_DIMS, NB_CLUSTERS,
                                  &alpha, GPU_centroid_T, NB_CLUSTERS,
                                  &beta, NULL, NB_DIMS,
                                  GPU_centroid, NB_DIMS), 
                      "Use CUBLAS_GEAM to transpose GPU_centroid_T");

  CHECK_CUDA_SUCCESS(hipMemcpy(centroid, GPU_centroid,
                               sizeof(T_real)*NB_CLUSTERS*NB_DIMS,
                               hipMemcpyDeviceToHost),
                    "Transfer centroids...");
}


/*-------------------------------------------------------------------------------*/
/* Initialize the random generator used for each centroid                        */
/*-------------------------------------------------------------------------------*/
__global__ void kernel_SetupcuRand(hiprandState *state)
{
  int centroidIdx = threadIdx.x + blockIdx.x * blockDim.x;

  if (centroidIdx < NB_CLUSTERS) {
    hiprand_init(4321, centroidIdx, 0, &state[centroidIdx]);
  }
}


/*-------------------------------------------------------------------------------*/
/* Select the initial centroids (uniformly at random) from the input data        */
/*-------------------------------------------------------------------------------*/
__global__ void kernel_InitializeCentroids(hiprandState *state, T_real *GPU_centroid_T, T_real *GPU_instance_T)
                                           /*T_real *GPU_centroid OR *GPU_centroid_T*/ 
                                           /*T_real *GPU_instance OR *GPU_instance_T*/
{
  int centroidIdx = threadIdx.x + blockIdx.x * blockDim.x;

  // Each initial centroid will be one of the input data 
  if (centroidIdx < NB_CLUSTERS) {
  
    // Get the current state of the random generator of the centroid
    hiprandState localState = state[centroidIdx];
    
    // Compute an idx value in [0, NB_INSTANCES - 1]: select an input data
    // Note: hiprand_uniform() returns a pseudo-random float in the range [0.0, 1.0[
    int idx = floor(NB_INSTANCES * CURAND_UNIFORM(&localState));
    
    // Set the centroid coordinates with the selected input data coordinates 
    for (int j = 0; j < NB_DIMS; j++)                                   // TO DO
    //  GPU_centroid[...] = GPU_instance[...]
    //  or GPU_centroid_T[...] = GPU_instance_T[...]
    //  or GPU_centroid[...] = GPU_instance_T[...]
    //  or GPU_centroid_T[...] = GPU_instance[...]

      // SHARED MEMORY TO IMPROVE? (should use Tranpose matrix of GPU_intance instead)
      // GPU_centroid_T[j][centroidIdx] = GPU_instance[idx][j];
      // GPU_centroid_T[j * NB_CLUSTERS + centroidIdx] = GPU_instance[idx * NB_DIMS + j];
      GPU_centroid_T[j * NB_CLUSTERS + centroidIdx] = GPU_instance_T[j * NB_INSTANCES + idx];

  }
}


/*-------------------------------------------------------------------------------*/
/* Compute distances and Assign each point to its nearest centorid               */
/*-------------------------------------------------------------------------------*/
__global__ void kernel_ComputeAssign(T_real *GPU_instance_T, T_real *GPU_centroid_T, int *GPU_label, unsigned long long *AdrGPU_change_total)
{
  // TO DO

  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int closest_centroid_idx = 0;
  T_real min_dist = REAL_MAX;

  if (idx < NB_INSTANCES) {
    for (int i = 0; i < NB_CLUSTERS; ++i) {
      T_real distance = 0.0;
      for (int j = 0; j < NB_DIMS; ++j) {
        T_real temp = (GPU_instance_T[j * NB_INSTANCES + idx] - GPU_centroid_T[j * NB_CLUSTERS + i]);
        distance += temp*temp;
      }
      if (distance < min_dist) {
        min_dist = distance;
        closest_centroid_idx = i;
      }
    }
    if (GPU_label[idx] != closest_centroid_idx) {
      atomicAdd(AdrGPU_change_total, 1);
      GPU_label[idx] = closest_centroid_idx;
    }
  }
 
  // Note:
  //   The "atomic add" on global GPU var could be useful:
  //     atomicAdd(Adr_of_GPU_var, Integer_Value_to_Add);
  //   Warning: time consumming function
}


__global__ void OLD_kernel_UpdateCentroid_Step1(T_real *GPU_instance_T, T_real *GPU_centroid_T, int *GPU_label, int *GPU_count)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  __shared__ T_real sh_instance[BLOCK_SIZE_X_N];
  __shared__ T_real sh_count[BLOCK_SIZE_X_N];
  

  if (idx < NB_INSTANCES) {
    for(int clusterIdx = 0; clusterIdx < NB_CLUSTERS; ++clusterIdx){
      for(int dim = 0; dim < NB_DIMS; ++dim) {
        if (GPU_label[idx] == clusterIdx) {
          sh_instance[threadIdx.x] = GPU_instance_T[dim * NB_INSTANCES + idx];
          sh_count[threadIdx.x] = 1;
        }
        else { // Divergence to test
          sh_instance[threadIdx.x] = 0;
          sh_count[threadIdx.x] = 0;
        }
        
        __syncthreads();

        // reduction : 
        // 1st step : half the threads should work
        for (unsigned int s = blockDim.x/2; s > 0; s >>= 1) {
          if (threadIdx.x < s) {
            sh_instance[threadIdx.x] += sh_instance[threadIdx.x + s];

            // Better way?
            if(dim == 0) {
              sh_count[threadIdx.x] += sh_count[threadIdx.x + s];
            }

          }
          __syncthreads();
        }

        // atomic add in GPU_centroid_T and GPU_count
        if (threadIdx.x == 0) {
          atomicAdd(&GPU_centroid_T[dim * NB_CLUSTERS + clusterIdx], sh_instance[0]);
          if (dim == 0) {
            atomicAdd(&GPU_count[clusterIdx], sh_count[0]);
          }
        }
      } 
    }
  }
}

__global__ void OLD2_kernel_UpdateCentroid_Step1(T_real *GPU_instance_T, T_real *GPU_centroid_T, int *GPU_label, int *GPU_count)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int dim = blockIdx.y;

  __shared__ T_real sh_instance[BLOCK_SIZE_X_N];
  __shared__ T_real sh_count[BLOCK_SIZE_X_N];

  if (idx < NB_INSTANCES) {
    for(int clusterIdx = 0; clusterIdx < NB_CLUSTERS; ++clusterIdx){
      // ?
      if (GPU_label[idx] == clusterIdx) {
        sh_count[threadIdx.x] = 1;
        sh_instance[threadIdx.x] = GPU_instance_T[NB_INSTANCES*dim + idx];
      }
      else {
        sh_count[threadIdx.x] = 0;
        sh_instance[threadIdx.x] = 0;
      }

      __syncthreads();

      // reduction : 
      // 1st step : half the threads should work
      for (unsigned int s = blockDim.x/2; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
          sh_instance[threadIdx.x] += sh_instance[threadIdx.x + s];
          sh_count[threadIdx.x] += sh_count[threadIdx.x + s];
        }
        __syncthreads();
      }

        // atomic add in GPU_centroid_T and GPU_count
      if (threadIdx.x == 0) {
        atomicAdd(&GPU_centroid_T[dim * NB_CLUSTERS + clusterIdx], sh_instance[0]);
        if (dim == 0) {
          atomicAdd(&GPU_count[clusterIdx], sh_count[0]);
        }
      }
    }
  }
}

__global__ void OLD3_kernel_UpdateCentroid_Step1(T_real *GPU_instance_T, T_real *GPU_centroid_T, int *GPU_label, int *GPU_count)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int dim = blockIdx.y;
  int clusterIdx = blockIdx.z;

  // if (threadIdx.x == 0) {
  //   printf("idx = %d, dim = %d\n", idx, dim);
  // }

  __shared__ T_real sh_instance[BLOCK_SIZE_X_N];
  __shared__ T_real sh_count[BLOCK_SIZE_X_N];

  if (idx < NB_INSTANCES) {
    if (GPU_label[idx] == clusterIdx) {
      sh_count[threadIdx.x] = 1;
      sh_instance[threadIdx.x] = GPU_instance_T[NB_INSTANCES*dim + idx];
    }
    else {
      sh_count[threadIdx.x] = 0;
      sh_instance[threadIdx.x] = 0;
    }


    // reduction : 
    // 1st step : half the threads should work
    // for (unsigned int s = blockDim.x/2; s > 0; s >>= 1) {
    //   if (threadIdx.x < s) {
    //     sh_instance[threadIdx.x] += sh_instance[threadIdx.x + s];
    //     sh_count[threadIdx.x] += sh_count[threadIdx.x + s];
    //   }
    //   __syncthreads();
    // }

    // loop unroll version :
    #if BLOCK_SIZE_X_N > 512
    __syncthreads();
    if (threadIdx.x < 512) {
      sh_instance[threadIdx.x] += sh_instance[threadIdx.x + 512];
      sh_count[threadIdx.x] += sh_count[threadIdx.x + 512];
    }
    else
    {
      return;
    }
    #endif
    
   

    #if BLOCK_SIZE_X_N > 256
    __syncthreads();
    if (threadIdx.x < 256) {
      sh_instance[threadIdx.x] += sh_instance[threadIdx.x + 256];
      sh_count[threadIdx.x] += sh_count[threadIdx.x + 256];
    }
    else {
      return;
    }
    #endif

    #if BLOCK_SIZE_X_N > 128
    __syncthreads();
    if (threadIdx.x < 128) {
      sh_instance[threadIdx.x] += sh_instance[threadIdx.x + 128];
      sh_count[threadIdx.x] += sh_count[threadIdx.x + 128];
    }
    else {
      return;
    }
    #endif

    #if BLOCK_SIZE_X_N > 64
    __syncthreads();
    if (threadIdx.x < 64) {
      sh_instance[threadIdx.x] += sh_instance[threadIdx.x + 64];
      sh_count[threadIdx.x] += sh_count[threadIdx.x + 64];
    }
    else {
      return;
    }
    #endif

    #if BLOCK_SIZE_X_N > 32
    __syncthreads();
    if (threadIdx.x < 32) {
      sh_instance[threadIdx.x] += sh_instance[threadIdx.x + 32];
      sh_count[threadIdx.x] += sh_count[threadIdx.x + 32];
    }
    else {
      return;
    }
    #endif

    #if BLOCK_SIZE_X_N > 16
    sh_instance[threadIdx.x] += sh_instance[threadIdx.x + 16];
    sh_count[threadIdx.x] += sh_count[threadIdx.x + 16];
    #endif

    #if BLOCK_SIZE_X_N > 8
    sh_instance[threadIdx.x] += sh_instance[threadIdx.x + 8];
    sh_count[threadIdx.x] += sh_count[threadIdx.x + 8];
    #endif

    #if BLOCK_SIZE_X_N > 4
    sh_instance[threadIdx.x] += sh_instance[threadIdx.x + 4];
    sh_count[threadIdx.x] += sh_count[threadIdx.x + 4];
    #endif

    #if BLOCK_SIZE_X_N > 2
    sh_instance[threadIdx.x] += sh_instance[threadIdx.x + 2];
    sh_count[threadIdx.x] += sh_count[threadIdx.x + 2];
    #endif

    #if BLOCK_SIZE_X_N > 1
    sh_instance[threadIdx.x] += sh_instance[threadIdx.x + 1];
    sh_count[threadIdx.x] += sh_count[threadIdx.x + 1];
    #endif

    // atomic add in GPU_centroid_T and GPU_count
    if (threadIdx.x == 0) {
      atomicAdd(&GPU_centroid_T[dim * NB_CLUSTERS + clusterIdx], sh_instance[0]);
      if (dim == 0) {
        atomicAdd(&GPU_count[clusterIdx], sh_count[0]);
      }
    }
  }
}

// Version 3 : no cluster loop
__global__ void kernel_UpdateCentroid_Step1(T_real *GPU_instance_T, T_real *GPU_centroid_T, int *GPU_label, int *GPU_count)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int dim = blockIdx.y;
  int clusterIdx = blockIdx.z;

  // if (threadIdx.x == 0) {
  //   printf("idx = %d, dim = %d\n", idx, dim);
  // }

  __shared__ T_real sh_instance[BLOCK_SIZE_X_N];
  __shared__ T_real sh_count[BLOCK_SIZE_X_N];

  if (idx < NB_INSTANCES) {
    if (GPU_label[idx] == clusterIdx) {
      sh_count[threadIdx.x] = 1;
      sh_instance[threadIdx.x] = GPU_instance_T[NB_INSTANCES*dim + idx];
    }
    else {
      sh_count[threadIdx.x] = 0;
      sh_instance[threadIdx.x] = 0;
    }

    __syncthreads();

    // reduction : 
    // 1st step : half the threads should work
    for (unsigned int s = blockDim.x/2; s > 0; s >>= 1) {
      if (threadIdx.x < s) {
        sh_instance[threadIdx.x] += sh_instance[threadIdx.x + s];
        sh_count[threadIdx.x] += sh_count[threadIdx.x + s];
      }
      __syncthreads();
    }

      // atomic add in GPU_centroid_T and GPU_count
    if (threadIdx.x == 0) {
      atomicAdd(&GPU_centroid_T[dim * NB_CLUSTERS + clusterIdx], sh_instance[0]);
      if (dim == 0) {
        atomicAdd(&GPU_count[clusterIdx], sh_count[0]);
      }
    }
  }
}

/*-------------------------------------------------------------------------------*/
/* Update centroids - step 2                                                     */
/*-------------------------------------------------------------------------------*/
__global__ void kernel_UpdateCentroid_Step2(T_real *GPU_centroid_T, int *GPU_count, T_real *GPU_instance_T, int *GPU_failed)
{

  int clusterIdx = threadIdx.x + blockIdx.x * blockDim.x;
  bool failed = false;
  
  if (clusterIdx < NB_CLUSTERS) {   
    int count = GPU_count[clusterIdx];
    if (count > 0) {
      for (int dim = 0; dim < NB_DIMS; ++dim) { 
        GPU_centroid_T[dim * NB_CLUSTERS + clusterIdx] /= count;   
      }
    }
    else {
      failed = true;
    }

    if (failed){
      atomicAdd(GPU_failed, 1);
    }
  }
 // TO DO

 // Note:
 //   The "atomic add" on global GPU var could be useful:
 //     atomicAdd(Adr_of_GPU_var, Integer_Value_to_Add);
 //   Warning: time consumming function
}


/*-------------------------------------------------------------------------------*/
/* Complete clustering on GPU, with loop control on CPU                          */
/*-------------------------------------------------------------------------------*/
void gpu_Kmeans()
{
  // End criteria variables
  double tolerance = 0.0;          // tolerance will be: nb_changes / NB_INSTANCES
  int nb_iter_kmeans = 0;

  dim3 Dg, Db;
  
  // Reset the array of labels (result of the clustering) to 0
  CHECK_CUDA_SUCCESS(hipMemset(GPU_label, 0, sizeof(int)*NB_INSTANCES), 
                     "Reset GPU_label to zeros");

  // Initialize the random generator used for each centroid
  Db.x = BLOCK_SIZE_X_C;
  Db.y = 1;
  Db.z = 1;
  Dg.x = NB_CLUSTERS/Db.x + (NB_CLUSTERS%Db.x > 0 ? 1 : 0);
  Dg.y = 1;
  Dg.z = 1;
  kernel_SetupcuRand<<<Dg,Db>>>(devStates);
  
  // Select initial centroids at random                              // TO DO
  // CudaCheckError();
  kernel_InitializeCentroids<<<Dg,Db>>>(devStates, GPU_centroid_T, GPU_instance_T);

  //CudaCheckError();

  // Note: IF NEEDED you can transpose a 2D array using CUBLAS_GEAM() function
  // Ex: Transpose GPU_centroid_T to GPU_centroid
  //
  //T_real alpha = 1.0f;
  //T_real beta = 0.0f;
  //CHECK_CUBLAS_SUCCESS(CUBLAS_GEAM(cublasHandle,
  //                                 HIPBLAS_OP_T, HIPBLAS_OP_N,
  //                                 NB_DIMS, NB_CLUSTERS,
  //                                 &alpha, GPU_centroid_T, NB_CLUSTERS,
  //                                 &beta, NULL, NB_DIMS,
  //                                 GPU_centroid, NB_DIMS), 
  //                     "Use CUBLAS_GEAM to transpose GPU_centroid_T");

  // Clustering iterative loop --------------------------------------------
  do {
    // - Reset the GPU counter of label changes at the current iteration
    CHECK_CUDA_SUCCESS(hipMemset(AdrGPU_change_total, 0, 
                                  sizeof(unsigned long long int)*1), 
                       "Reset GPU_change_total to zero");

    // - Compute distance & Assign points to clusters 
    Db.x = BLOCK_SIZE_X_N;
    Db.y = 1;
    Db.z = 1;
    Dg.x = NB_INSTANCES/Db.x + (NB_INSTANCES%Db.x > 0 ? 1 : 0);
    Dg.y = 1;
    Dg.z = 1;
    kernel_ComputeAssign<<<Dg,Db>>>(GPU_instance_T, GPU_centroid_T, GPU_label, AdrGPU_change_total);
    CudaCheckError();

    CHECK_CUDA_SUCCESS(hipMemcpy(&nb_changes, AdrGPU_change_total, 
                                  sizeof(unsigned long long int)*1, 
                                  hipMemcpyDeviceToHost),
                       "Transfer GPU_change_total-->nb_changes");


    // - Update Centroids - step 1
    // -- reset the array of counters of points associated to each cluster
    //CHECK_CUDA_SUCCESS(hipMemset(GPU_count, 0,...), "Reset GPU_count to zeros");
    // -- reset the array of centroid coordinates 
    //CHECK_CUDA_SUCCESS(hipMemset(..., ..., ...), "Reset GPU centroids");
    
    // -- compute the number of points associated to each cluster
    //   and compute the sum of their coordinates (to compute their barycenter in next kernel)
    //   Note : you can use atomicAdd(...) ... and shared memory to reduce the nb of atomicAdd....
    // Db.x = BLOCK_SIZE_X_N;
    // Db.y = 1;
    // Db.z = 1;
    // Dg.x = NB_INSTANCES/Db.x + (NB_INSTANCES%Db.x > 0 ? 1 : 0);
    // Dg.y = 1;
    // Dg.z = 1;


    // Db.x = BLOCK_SIZE_X_N;
    // Db.y = 1;
    // Db.z = 1;
    // Dg.x = NB_INSTANCES/Db.x + (NB_INSTANCES%Db.x > 0 ? 1 : 0);
    // Dg.y = NB_DIMS; // Not in Db.y to have different shared mem for different dim
    // Dg.z = 1;

    // printf("Before grille setup\n");

    Db.x = BLOCK_SIZE_X_N;
    
    Db.y = 1;
    Db.z = 1;
    Dg.x = NB_INSTANCES/Db.x + (NB_INSTANCES%Db.x > 0 ? 1 : 0);
    // printf("Before grille setup\n");
    Dg.y = NB_DIMS; 
    // printf("Before grille setup\n");
    // printf("NB_CLUSTERS = %d\n", NB_CLUSTERS);
    Dg.z = 4;
    // printf("hardcoded\n");
    Dg.z = NB_CLUSTERS;
    // printf("Before comm\n");
    
    // printf("NB_DIMS = %d, NB_INSTANCES = %d\n", NB_DIMS, NB_INSTANCES);
    // printf("Db.x = %d, Db.y = %d, Db.z = %d\n", Db.x, Db.y, Db.z);

    // printf("Dg.z");

    // TO verify
    // intialize GPU_centroid_T and GPU_count to 0
    CHECK_CUDA_SUCCESS(hipMemset(GPU_centroid_T, 0, sizeof(T_real)*NB_DIMS*NB_CLUSTERS), "Reset GPU_centroid_T to zeros");
    CHECK_CUDA_SUCCESS(hipMemset(GPU_count, 0, sizeof(int)*NB_CLUSTERS), "Reset GPU_count to zeros");
    
    // printf("Im here2");

    kernel_UpdateCentroid_Step1<<<Dg,Db>>>(GPU_instance_T, GPU_centroid_T, GPU_label, GPU_count);



    // Db.x = BLOCK_SIZE_X_C;
    // Db.y = 1;
    // Db.z = 1;
    // Dg.x = NB_CLUSTERS/Db.x + (NB_CLUSTERS%Db.x > 0 ? 1 : 0);
    // Dg.y = 1;
    // Dg.z = 1;
    // kernel_UpdateCentroid_Step2<<<Dg,Db>>>(GPU_centroid_T, GPU_count, GPU_instance_T);

    // Initialize the random generator used for each centroid
    
    

    Db.x = BLOCK_SIZE_X_C;
    Db.y = 1;
    Db.z = 1;
    Dg.x = NB_CLUSTERS/Db.x + (NB_CLUSTERS%Db.x > 0 ? 1 : 0);
    Dg.y = 1;
    Dg.z = 1;

    CHECK_CUDA_SUCCESS(hipMemset(GPU_failed, 0, sizeof(int)*1), "Reset GPU_failed to zeros");

    kernel_UpdateCentroid_Step2<<<Dg,Db>>>(GPU_centroid_T, GPU_count, GPU_instance_T, GPU_failed);
    
    int failed = 0;

    CHECK_CUDA_SUCCESS(hipMemcpy(&failed, GPU_failed, 
      sizeof(int)*1, 
      hipMemcpyDeviceToHost),
      "Transfer labels 'failed'...");

    if (failed>0) {
      CHECK_CUDA_SUCCESS(hipMemset(GPU_label, 0, sizeof(int)*NB_INSTANCES), 
                     "Reset GPU_label to zeros");


      kernel_SetupcuRand<<<Dg,Db>>>(devStates);
      
      // Select initial centroids at random                              // TO DO
      // CudaCheckError();
      kernel_InitializeCentroids<<<Dg,Db>>>(devStates, GPU_centroid_T, GPU_instance_T);
    }

    CudaCheckError();

    // - End criteria computation
    tolerance = ((double)nb_changes) / NB_INSTANCES;     
    printf("Track = %llu  Tolerance = %lf\n", nb_changes, tolerance); 
    nb_iter_kmeans++;
    
  } while (tolerance > TOL_KMEANS && nb_iter_kmeans < MAX_ITER_KMEANS);

  // To measure correct time in main.cc
  //hipDeviceSynchronize();   // not necessary if you call CudaCheckError() 
                               // that already wait the end of last GPU op.
}

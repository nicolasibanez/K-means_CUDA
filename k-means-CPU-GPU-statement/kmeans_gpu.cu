#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// k-means on CPU & GPU
// S. Vialle March 2022 (with the help of G. He)
////////////////////////////////////////////////////////////////////////////////

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h> 
#include <hip/hip_runtime.h>
#include <hipblas.h>              // for CUBLAS_GEAM
#include <hiprand/hiprand_kernel.h>          // for CURAND_UNIFORM
#include <float.h>

#include "main.h"
#include "init.h"
#include "kmeans_gpu.h"


/*----------------------------------------------------------------------------*/
/* Define pointers on GPU variables, and GPU symbols                          */
/*----------------------------------------------------------------------------*/
// Choose the ONE you need (big array)
T_real *GPU_instance;      //[NB_INSTANCES][NB_DIMS] --> [NB_INSTANCES * NB_DIMS]
T_real *GPU_instance_T;    //[NB_DIMS][NB_INSTANCES] --> [NB_DIMS * NB_INSTANCES]

// Choose the one you need or both (small arrays)
T_real *GPU_centroid;      //[NB_CLUSTERS][NB_DIMS] --> [NB_CLUSTERS * NB_DIMS]
T_real *GPU_centroid_T;    //[NB_DIMS][NB_CLUSTERS] --> [NB_DIMS * NB_CLUSTERS]

int *GPU_label;             //[NB_INSTANCES] Label of each point
int *GPU_change;            //[NB_INSTANCES] Flag recording the change of label
int *GPU_count;             //[NB_CLUSTERS]  Count of instance points in each cluster
int *GPU_failed;

__device__ unsigned long long GPU_change_total; // nb of label changes at current iter
unsigned long long *AdrGPU_change_total = NULL;

hiprandState *devStates;     // To use hiprand
hipblasHandle_t cublasHandle;// To activate cublas (speedup the GPU!)


/*-------------------------------------------------------------------------------*/
/* Init and finalize the GPU device.                                             */
/*-------------------------------------------------------------------------------*/
void gpu_Init()
{
  hipInit(0);

  // Allocate memory space for GPU arrays
  
  // Choose ONE (Big array)
  CHECK_CUDA_SUCCESS(hipMalloc((void**) &GPU_instance, sizeof(T_real)*NB_INSTANCES*NB_DIMS), "Dynamic allocation for GPU_instance");
  CHECK_CUDA_SUCCESS(hipMalloc((void**) &GPU_instance_T, sizeof(T_real)*NB_DIMS*NB_INSTANCES), "Dynamic allocation for GPU_instance_T");
  

  // Choose one or both (small array)
  CHECK_CUDA_SUCCESS(hipMalloc((void**) &GPU_centroid, sizeof(T_real)*NB_CLUSTERS*NB_DIMS), "Dynamic allocation for GPU_centroid");
  CHECK_CUDA_SUCCESS(hipMalloc((void**) &GPU_centroid_T, sizeof(T_real)*NB_DIMS*NB_CLUSTERS), "Dynamic allocation for GPU_centroid_T");
  
  CHECK_CUDA_SUCCESS(hipMalloc((void**) &GPU_label, sizeof(int)*NB_INSTANCES), "Dynamic allocation for GPU_label");
  CHECK_CUDA_SUCCESS(hipMalloc((void**) &GPU_change, sizeof(int)*NB_INSTANCES), "Dynamic allocation for GPU_change");
  CHECK_CUDA_SUCCESS(hipMalloc((void**) &GPU_count, sizeof(int)*NB_CLUSTERS), "Dynamic allocation for GPU_count");

  CHECK_CUDA_SUCCESS(hipMalloc((void**) &GPU_failed, sizeof(int)*1), "Dynamic allocation for GPU_failed");
 
  // Initialize an array of "hiprandState) for using hiprand
  CHECK_CUDA_SUCCESS(hipMalloc((void**) &devStates, sizeof(hiprandState)*NB_CLUSTERS), "Dynamic allocation for devStates");

  // Get address of GPU symbols
  CHECK_CUDA_SUCCESS(hipGetSymbolAddress((void **) &AdrGPU_change_total, GPU_change_total), 
                     "Get the address of GPU_change_total");
                     
  // Initialize CUBLAS lib usage
  CHECK_CUBLAS_SUCCESS(hipblasCreate(&cublasHandle), "Init of the CUBLAS lib handle"); 
}

void gpu_Finalize()
{
  // Free dynamic allocations (function of the arrays you used)
  CHECK_CUDA_SUCCESS(hipFree(GPU_instance), "Free the dynamic allocation for GPU_instance");
  CHECK_CUDA_SUCCESS(hipFree(GPU_instance_T), "Free the dynamic allocation for GPU_instance_T");

  CHECK_CUDA_SUCCESS(hipFree(GPU_centroid), "Free the dynamic allocation for GPU_centroid");
  CHECK_CUDA_SUCCESS(hipFree(GPU_centroid_T), "Free the dynamic allocation for GPU_centroid_T");
  CHECK_CUDA_SUCCESS(hipFree(GPU_label), "Free the dynamic allocation for GPU_label");
  CHECK_CUDA_SUCCESS(hipFree(GPU_change), "Free the dynamic allocation for GPU_change");
  CHECK_CUDA_SUCCESS(hipFree(GPU_count), "Free the dynamic allocation for GPU_count");

  CHECK_CUDA_SUCCESS(hipFree(GPU_failed), "Free the dynamic allocation for GPU_failed");
  
  // Free array of curandStates
  CHECK_CUDA_SUCCESS(hipFree(devStates), "Free the dynamic allocation for devStates");

  // Free CUBLAS lib usage
  CHECK_CUBLAS_SUCCESS(hipblasDestroy(cublasHandle), "Free the CUBLAS lib");
}


/*-------------------------------------------------------------------------------*/
/* Transfer of CPU input data into GPU symbols                                   */
/*-------------------------------------------------------------------------------*/
void gpu_SetDataOnGPU()
{
  // Transfer instance[] or instance_T[] .... as you want                  // TO DO
  CHECK_CUDA_SUCCESS(hipMemcpy(GPU_instance, instance,
                                sizeof(T_real)*NB_DIMS*NB_INSTANCES, 
                                hipMemcpyHostToDevice),
                      "Transfer instance...");


  
  T_real alpha = 1.0f;
  T_real beta = 0.0f;
  CHECK_CUBLAS_SUCCESS(CUBLAS_GEAM(cublasHandle,
                                  HIPBLAS_OP_T, HIPBLAS_OP_N,
                                  NB_INSTANCES, NB_DIMS,
                                  &alpha, GPU_instance, NB_DIMS,
                                  &beta, NULL, NB_INSTANCES,
                                  GPU_instance_T, NB_INSTANCES), 
                      "Use CUBLAS_GEAM to transpose GPU_instance");
}


/*-------------------------------------------------------------------------------*/
/* Transfer of GPU results into CPU array                                        */
/*-------------------------------------------------------------------------------*/
void gpu_GetResultOnCPU()
{
  // Transfer labels computed on GPU, to the CPU                           // TO DO
  CHECK_CUDA_SUCCESS(hipMemcpy(label, GPU_label, 
                               sizeof(int)*NB_INSTANCES, 
                               hipMemcpyDeviceToHost),
                    "Transfer labels...");

  // Transfer final centroids computed on GPU, to the CPU
  T_real alpha = 1.0f;
  T_real beta = 0.0f;
  CHECK_CUBLAS_SUCCESS(CUBLAS_GEAM(cublasHandle,
                                  HIPBLAS_OP_T, HIPBLAS_OP_N,
                                  NB_DIMS, NB_CLUSTERS,
                                  &alpha, GPU_centroid_T, NB_CLUSTERS,
                                  &beta, NULL, NB_DIMS,
                                  GPU_centroid, NB_DIMS), 
                      "Use CUBLAS_GEAM to transpose GPU_centroid_T");

  CHECK_CUDA_SUCCESS(hipMemcpy(centroid, GPU_centroid,
                               sizeof(T_real)*NB_CLUSTERS*NB_DIMS,
                               hipMemcpyDeviceToHost),
                    "Transfer centroids...");
}


/*-------------------------------------------------------------------------------*/
/* Initialize the random generator used for each centroid                        */
/*-------------------------------------------------------------------------------*/
__global__ void kernel_SetupcuRand(hiprandState *state)
{
  int centroidIdx = threadIdx.x + blockIdx.x * blockDim.x;

  if (centroidIdx < NB_CLUSTERS) {
    hiprand_init(4321, centroidIdx, 0, &state[centroidIdx]);
  }
}


/*-------------------------------------------------------------------------------*/
/* Select the initial centroids (uniformly at random) from the input data        */
/*-------------------------------------------------------------------------------*/
__global__ void kernel_InitializeCentroids(hiprandState *state, T_real *GPU_centroid_T, T_real *GPU_instance_T)
{
  int centroidIdx = threadIdx.x + blockIdx.x * blockDim.x;

  // Each initial centroid will be one of the input data 
  if (centroidIdx < NB_CLUSTERS) {
  
    // Get the current state of the random generator of the centroid
    hiprandState localState = state[centroidIdx];
    
    // Compute an idx value in [0, NB_INSTANCES - 1]: select an input data
    // Note: hiprand_uniform() returns a pseudo-random float in the range [0.0, 1.0[
    int idx = floor(NB_INSTANCES * CURAND_UNIFORM(&localState));
    
    // Set the centroid coordinates with the selected input data coordinates 
    for (int j = 0; j < NB_DIMS; j++)
      GPU_centroid_T[j * NB_CLUSTERS + centroidIdx] = GPU_instance_T[j * NB_INSTANCES + idx];

  }
}




/*-------------------------------------------------------------------------------*/
/* Compute distances and Assign each point to its nearest centorid               */
/*-------------------------------------------------------------------------------*/
__global__ void kernel_ComputeAssign(T_real *GPU_instance_T, T_real *GPU_centroid_T, int *GPU_label, unsigned long long *AdrGPU_change_total)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int closest_centroid_idx = 0;
  T_real min_dist = REAL_MAX;

  // 1d instead :
  __shared__ T_real sh_centroid_T[NB_DIMS*NB_CLUSTERS];
  __shared__ T_real sh_instance_T[NB_DIMS*BLOCK_SIZE_X_N];

  // load centroids in shared memory
  if (threadIdx.x < NB_CLUSTERS) {
    for (int j = 0; j < NB_DIMS; ++j) {
      sh_centroid_T[j * NB_CLUSTERS + threadIdx.x] = GPU_centroid_T[j * NB_CLUSTERS + threadIdx.x];
    }
  }

  if (idx < NB_INSTANCES) {
    for (int j = 0; j < NB_DIMS; ++j) {
      sh_instance_T[j * BLOCK_SIZE_X_N + threadIdx.x] = GPU_instance_T[j * NB_INSTANCES + idx];
    }
    __syncthreads();

    for (int i = 0; i < NB_CLUSTERS; ++i) {
      T_real distance = 0.0;
      for (int j = 0; j < NB_DIMS; ++j) {
        T_real temp = (sh_instance_T[j * BLOCK_SIZE_X_N + threadIdx.x] - sh_centroid_T[j * NB_CLUSTERS + i]);
        distance += temp*temp;
      }


      if (distance < min_dist) {
        min_dist = distance;
        closest_centroid_idx = i;
      }
    }

    if (GPU_label[idx] != closest_centroid_idx) {
      atomicAdd(AdrGPU_change_total, 1);
      GPU_label[idx] = closest_centroid_idx;
    }
  }
}

/*-------------------------------------------------------------------------------*/
/* Update centroids - step 1                                                     */
/*-------------------------------------------------------------------------------*/
__global__ void kernel_UpdateCentroid_Step1(T_real *GPU_instance_T, T_real *GPU_centroid_T, int *GPU_label, int *GPU_count)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int dim = blockIdx.y;
  int clusterIdx = blockIdx.z;

  __shared__ T_real sh_instance[BLOCK_SIZE_X_N];
  __shared__ T_real sh_count[BLOCK_SIZE_X_N];

  sh_count[threadIdx.x] = 0;
  sh_instance[threadIdx.x] = 0;

  if (idx < NB_INSTANCES) {
    if (GPU_label[idx] == clusterIdx) {
      sh_count[threadIdx.x] = 1;
      sh_instance[threadIdx.x] = GPU_instance_T[NB_INSTANCES*dim + idx];
    }

    #if BLOCK_SIZE_X_N > 1024
    __syncthreads();
    if (threadIdx.x < 1024) {
      sh_instance[threadIdx.x] += sh_instance[threadIdx.x + 1024];
      sh_count[threadIdx.x] += sh_count[threadIdx.x + 1024];
    }
    else
    {
      return;
    }
    #endif


    #if BLOCK_SIZE_X_N > 512
    __syncthreads();
    if (threadIdx.x < 512) {
      sh_instance[threadIdx.x] += sh_instance[threadIdx.x + 512];
      sh_count[threadIdx.x] += sh_count[threadIdx.x + 512];
    }
    else
    {
      return;
    }
    #endif

    #if BLOCK_SIZE_X_N > 256
    __syncthreads();
    if (threadIdx.x < 256) {
      sh_instance[threadIdx.x] += sh_instance[threadIdx.x + 256];
      sh_count[threadIdx.x] += sh_count[threadIdx.x + 256];
    }
    else {
      return;
    }
    #endif

    #if BLOCK_SIZE_X_N > 128
    __syncthreads();
    if (threadIdx.x < 128) {
      sh_instance[threadIdx.x] += sh_instance[threadIdx.x + 128];
      sh_count[threadIdx.x] += sh_count[threadIdx.x + 128];
    }
    else {
      return;
    }
    #endif

    #if BLOCK_SIZE_X_N > 64
    __syncthreads();
    if (threadIdx.x < 64) {
      sh_instance[threadIdx.x] += sh_instance[threadIdx.x + 64];
      sh_count[threadIdx.x] += sh_count[threadIdx.x + 64];
    }
    else {
      return;
    }
    #endif

    #if BLOCK_SIZE_X_N > 32
    __syncthreads();
    if (threadIdx.x < 32) {
      sh_instance[threadIdx.x] += sh_instance[threadIdx.x + 32];
      sh_count[threadIdx.x] += sh_count[threadIdx.x + 32];
    }
    else {
      return;
    }
    #endif

    #if BLOCK_SIZE_X_N > 16
    sh_instance[threadIdx.x] += sh_instance[threadIdx.x + 16];
    sh_count[threadIdx.x] += sh_count[threadIdx.x + 16];
    #endif

    #if BLOCK_SIZE_X_N > 8
    sh_instance[threadIdx.x] += sh_instance[threadIdx.x + 8];
    sh_count[threadIdx.x] += sh_count[threadIdx.x + 8];
    #endif

    #if BLOCK_SIZE_X_N > 4
    sh_instance[threadIdx.x] += sh_instance[threadIdx.x + 4];
    sh_count[threadIdx.x] += sh_count[threadIdx.x + 4];
    #endif

    #if BLOCK_SIZE_X_N > 2
    sh_instance[threadIdx.x] += sh_instance[threadIdx.x + 2];
    sh_count[threadIdx.x] += sh_count[threadIdx.x + 2];
    #endif

    #if BLOCK_SIZE_X_N > 1
    sh_instance[threadIdx.x] += sh_instance[threadIdx.x + 1];
    sh_count[threadIdx.x] += sh_count[threadIdx.x + 1];
    #endif

    if (threadIdx.x == 0 && sh_count[0] > 0) {
      atomicAdd(&GPU_centroid_T[dim * NB_CLUSTERS + clusterIdx], sh_instance[0]);
      if (dim == 0) {
        atomicAdd(&GPU_count[clusterIdx], sh_count[0]);
      }
    }
  }
}

/*-------------------------------------------------------------------------------*/
/* Update centroids - step 2                                                     */
/*-------------------------------------------------------------------------------*/
__global__ void kernel_UpdateCentroid_Step2(T_real *GPU_centroid_T, int *GPU_count, T_real *GPU_instance_T, int *GPU_failed)
{

  int clusterIdx = threadIdx.x + blockIdx.x * blockDim.x;
  bool failed = false;
  
  if (clusterIdx < NB_CLUSTERS) {   
    int count = GPU_count[clusterIdx];
    if (count > 0) {
      for (int dim = 0; dim < NB_DIMS; ++dim) { 
        GPU_centroid_T[dim * NB_CLUSTERS + clusterIdx] /= count;   
      }
    }
    else {
      failed = true;
    }

    if (failed){
      atomicAdd(GPU_failed, 1);
    }
  }
}


/*-------------------------------------------------------------------------------*/
/* Complete clustering on GPU, with loop control on CPU                          */
/*-------------------------------------------------------------------------------*/
void gpu_Kmeans()
{
  // End criteria variables
  double tolerance = 0.0;          // tolerance will be: nb_changes / NB_INSTANCES
  int nb_iter_kmeans = 0;

  dim3 Dg, Db;
  
  // Reset the array of labels (result of the clustering) to 0
  CHECK_CUDA_SUCCESS(hipMemset(GPU_label, 0, sizeof(int)*NB_INSTANCES), 
                     "Reset GPU_label to zeros");

  // Initialize the random generator used for each centroid
  Db.x = BLOCK_SIZE_X_C;
  Db.y = 1;
  Db.z = 1;
  Dg.x = NB_CLUSTERS/Db.x + (NB_CLUSTERS%Db.x > 0 ? 1 : 0);
  Dg.y = 1;
  Dg.z = 1;
  kernel_SetupcuRand<<<Dg,Db>>>(devStates);
  
  // Select initial centroids at random
  kernel_InitializeCentroids<<<Dg,Db>>>(devStates, GPU_centroid_T, GPU_instance_T);

  // Clustering iterative loop --------------------------------------------
  do {
    // - Reset the GPU counter of label changes at the current iteration
    CHECK_CUDA_SUCCESS(hipMemset(AdrGPU_change_total, 0, 
                                  sizeof(unsigned long long int)*1), 
                       "Reset GPU_change_total to zero");

    // - Compute distance & Assign points to clusters 
    Db.x = BLOCK_SIZE_X_N;
    Db.y = 1;
    Db.z = 1;
    Dg.x = NB_INSTANCES/Db.x + (NB_INSTANCES%Db.x > 0 ? 1 : 0);
    Dg.y = 1;
    Dg.z = 1;
    kernel_ComputeAssign<<<Dg,Db>>>(GPU_instance_T, GPU_centroid_T, GPU_label, AdrGPU_change_total);

    CHECK_CUDA_SUCCESS(hipMemcpy(&nb_changes, AdrGPU_change_total, 
                                  sizeof(unsigned long long int)*1, 
                                  hipMemcpyDeviceToHost),
                       "Transfer GPU_change_total-->nb_changes");


    // -- compute the number of points associated to each cluster
    Db.x = BLOCK_SIZE_X_N;
    Db.y = 1;
    Db.z = 1;
    Dg.x = NB_INSTANCES/Db.x + (NB_INSTANCES%Db.x > 0 ? 1 : 0);
    Dg.y = NB_DIMS;
    Dg.z = NB_CLUSTERS;

    // intialize GPU_centroid_T and GPU_count to 0
    CHECK_CUDA_SUCCESS(hipMemset(GPU_centroid_T, 0, sizeof(T_real)*NB_DIMS*NB_CLUSTERS), "Reset GPU_centroid_T to zeros");
    CHECK_CUDA_SUCCESS(hipMemset(GPU_count, 0, sizeof(int)*NB_CLUSTERS), "Reset GPU_count to zeros");

    kernel_UpdateCentroid_Step1<<<Dg,Db>>>(GPU_instance_T, GPU_centroid_T, GPU_label, GPU_count);


    Db.x = BLOCK_SIZE_X_C;
    Db.y = 1;
    Db.z = 1;
    Dg.x = NB_CLUSTERS/Db.x + (NB_CLUSTERS%Db.x > 0 ? 1 : 0);
    Dg.y = 1;
    Dg.z = 1;

    CHECK_CUDA_SUCCESS(hipMemset(GPU_failed, 0, sizeof(int)*1), "Reset GPU_failed to zeros");

    kernel_UpdateCentroid_Step2<<<Dg,Db>>>(GPU_centroid_T, GPU_count, GPU_instance_T, GPU_failed);
    
    int failed = 0;

    CHECK_CUDA_SUCCESS(hipMemcpy(&failed, GPU_failed, 
      sizeof(int)*1, 
      hipMemcpyDeviceToHost),
      "Transfer labels 'failed'...");

    if (failed>0) {
      CHECK_CUDA_SUCCESS(hipMemset(GPU_label, 0, sizeof(int)*NB_INSTANCES), 
                     "Reset GPU_label to zeros");


      kernel_SetupcuRand<<<Dg,Db>>>(devStates);
      
      // Re-initialize centroids at random
      kernel_InitializeCentroids<<<Dg,Db>>>(devStates, GPU_centroid_T, GPU_instance_T);
    }

    // CudaCheckError();

    // - End criteria computation
    tolerance = ((double)nb_changes) / NB_INSTANCES;     
    printf("Track = %llu  Tolerance = %lf\n", nb_changes, tolerance); 
    nb_iter_kmeans++;
    
  } while (tolerance > TOL_KMEANS && nb_iter_kmeans < MAX_ITER_KMEANS);

  // To measure correct time in main.cc
  hipDeviceSynchronize();   // not necessary if you call CudaCheckError() 
                             //  that already wait the end of last GPU op.
}
